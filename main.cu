#include "hip/hip_runtime.h"
// Remember to ask Charlie:
// 1. Can a CUDA helper function return anything?
// 2. MemCpy inside the kernel?
// 3. Removed the "delete;" lines from the classes (bitmap), still getting the C++ error;
// Need to hardcore num scene objects
// Change that vector to an array
// No For Each

#include <math.h>
#include <stdio.h>
#include <vector>
#include <pthread.h>
#include <SDL.h>

#include "bitmap.hh"
#include "geom.hh"
#include "gui.hh"
#include "util.hh"
#include "vec.hh"

#ifdef __HIPCC__
#define CUDA_CALLABLE_MEMBER __host__ __device__
#else
#define CUDA_CALLABLE_MEMBER
#endif

// CUDA error checking from http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-ap
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Screen size
#define WIDTH 640
#define HEIGHT 480
#define N WIDTH*HEIGHT

// Rendering Properties
#define AMBIENT 0.3         // Ambient illumination
#define OVERSAMPLE 2        // Sample 2x2 subpixels
#define MAX_REFLECTIONS 10  // The maximum number of times a ray is reflected
#define EPSILON 0.03        // Shift points off surfaces by this much
// Create threads for oversampling
#define num_over_thread OVERSAMPLE*OVERSAMPLE
#define THREADS_PER_BLOCK 8
#define OBJ_NUM 3
#define LIGHT_NUM 2
using namespace std;

// Set up the 3D scene
void init_scene();

// Trace a ray through the scene to determine its color
CUDA_CALLABLE_MEMBER vec raytrace(vec origin, vec dir, size_t reflections, sphere* gpu_scene);

// A list of shapes that make up the 3D scene. Initialized by init_scene
sphere scene[OBJ_NUM];

// A list of light positions, all emitting pure white light
vec lights[LIGHT_NUM];

// computes the color for the quadrants
__global__ void set_quadrant_color(viewport* view, vec* result_array, sphere* gpu_scene);

/**
 * Entry point for the raytracer
 * \param argc  The number of command line arguments
 * \param argv  An array of command line arguments
 */
int main(int argc, char** argv) {
  // Create a GUI window
  gui ui("Raytracer", WIDTH, HEIGHT);
  
  // Initialize the 3D scene
  init_scene();

  // GPU shapes
  sphere* gpu_spheres;
  if (hipMalloc(&gpu_spheres, sizeof(sphere) * OBJ_NUM) != hipSuccess) {
    fprintf( stderr, "Fail to allocate GPU objects\n");
  }
  if(hipMemcpy(gpu_spheres, scene, sizeof(sphere) * OBJ_NUM, hipMemcpyHostToDevice) != hipSuccess) {
    fprintf( stderr, "Fail to copy objects to GPU\n");
  }

    
  // GPU lights
 vec* gpu_lights;
if (hipMalloc(&gpu_lights, sizeof(vec) * LIGHT_NUM)!= hipSuccess) {
    fprintf( stderr, "Fail to allocate GPU lights\n");
  }
  if(hipMemcpy(gpu_lights, lights,sizeof(vec) * LIGHT_NUM, hipMemcpyHostToDevice) != hipSuccess) {
    fprintf( stderr, "Fail to copy lights to GPU\n");
  }
    
  
  // Set up the viewport
  viewport view(vec(0, 100, -300), // Look from here
                vec(0, -0.25, 1),  // Look in this direction
                vec(0, 1, 0),      // Up is up
                WIDTH,             // Use screen width
                HEIGHT);           // Use screen height
  
  // Save the starting time
  size_t start_time = time_ms();
  
  bool running = true;

  
  // Loop until we get a quit event
  while(running) {
    // Process events
    SDL_Event event;
    while(SDL_PollEvent(&event) == 1) {
      // If the event is a quit event, then leave the loop
      if(event.type == SDL_QUIT) running = false;
    }
    
    // Rotate the camera around the scene once every five seconds
    float yrot = (time_ms() - start_time)/5000.0 * M_PI * 2;
    
    // Render the frame to this bitmap
    bitmap cpu_bmp;
    bitmap* gpu_bmp;
    vec cpu_result_array[WIDTH][HEIGHT];
    vec* gpu_result_array;
  
    // Allocate memory for the gpu bitmap and the gpu result array
    gpuErrchk(hipMalloc(&gpu_bmp, cpu_bmp.size()));
    
    if (hipMalloc(&gpu_result_array, sizeof(vec) * WIDTH * HEIGHT)!= hipSuccess) {
      fprintf( stderr, "Fail to allocate GPU result_array\n");
    }

    // Copy memory from the cpu bitmap and result array to the gpu counterparts
    if(hipMemcpy(gpu_bmp, &cpu_bmp, cpu_bmp.size(), hipMemcpyHostToDevice) != hipSuccess) {
      fprintf( stderr, "Fail to copy bitmap to GPU\n");
    }
    // why are we copying from cpu_result array to gpu?
    if(hipMemcpy(gpu_result_array, cpu_result_array, sizeof(vec) * WIDTH * HEIGHT, hipMemcpyHostToDevice) != hipSuccess) {
      fprintf( stderr, "Fail to copy result_array to GPU\n");
    }

    // allocating necessary variables for raytrace

    // viewport
    viewport* gpu_viewport;
    if (hipMalloc(&gpu_viewport, sizeof(viewport))!= hipSuccess) {
      fprintf( stderr, "Fail to allocate GPU viewport\n");
    }
    if(hipMemcpy(gpu_viewport, &view, sizeof(viewport), hipMemcpyHostToDevice) != hipSuccess) {
      fprintf( stderr, "Fail to copy viewport to GPU\n");
    }

    // a thread for each pixel
    set_quadrant_color <<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK,
      THREADS_PER_BLOCK>>> (gpu_viewport, gpu_result_array, gpu_spheres);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    

    // copy result array to CPU
    if(hipMemcpy(cpu_result_array, gpu_result_array, sizeof(vec) * WIDTH * HEIGHT, hipMemcpyDeviceToHost)
       != hipSuccess) {
      fprintf( stderr, "Fail to copy result_array to CPU\n");
    }


    // would it be faster to do this inside the kernel, and then copy over the bitmap in the end?
    // instead of writing to an array and copying that back and then running these 2 for loops?
    for (int x = 0 ; x < WIDTH; x++){
      for(int y = 0; y < HEIGHT; y++){
        cpu_bmp.set(x, y, cpu_result_array[x][y]);
      }
    }

    // Display the rendered frame
    ui.display(cpu_bmp);
  }
  
  return 0;
}

// computes the color for the quadrants
__global__ void set_quadrant_color(viewport* view, vec* result_array, sphere* gpu_spheres){
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int index_x = index % WIDTH;
  int index_y = index / WIDTH;
  if(index_y >= HEIGHT || index_x >= WIDTH || index >= HEIGHT*WIDTH) {
    printf("%d, %d\n", index_x, index_y);
  }
  vec result = raytrace(view->origin(), view->dir(index_x, index_y), 0, gpu_spheres);
  //vec result = vec(AMBIENT, AMBIENT, AMBIENT);
  // Set the pixel color
  result_array[index] = result;
}

/**
 * Follow a ray backwards through the scene and return the ray's color
 * \param origin        The origin of the ray
 * \param dir           The direction of the ray
 * \param reflections   The number of times this ray has been reflected
 * \returns             The color of this ray
 */
CUDA_CALLABLE_MEMBER vec raytrace(vec origin, vec dir, size_t reflections, sphere* gpu_spheres) {
  
  // Normalize the direction vector
  dir = dir.normalized();
  
  // Keep track of the closest shape that is intersected by this ray
  sphere* intersected = NULL;
  float intersect_distance = 0;
  
  // Loop over all shapes in the scene to find the closest intersection
  for(int i = 0; i < OBJ_NUM; i++) {
    float distance = gpu_spheres[i].intersection(origin, dir);
    if(distance >= 0 && (distance < intersect_distance || intersected == NULL)) {
      intersect_distance = distance;
      intersected = &gpu_spheres[i];
    }
  }
  
  // If the ray didn't intersect anything, just return the ambient color
  if(intersected == NULL) return vec(AMBIENT, AMBIENT, AMBIENT);

  // Without reflections
  
  // Compute the point where the intersection occurred
  vec intersection = origin + dir * intersect_distance;
  
  // Otherwise just return the color of the object
  return intersected->get_color(intersection);

  // With reflections

  /*
  // The new starting point for the reflected ray is the point of intersection.
  // Find the reflection point just a *little* closer so it isn't on the object.
  // Otherwise, the new ray may intersect the same shape again depending on
  // rounding error.

  vec intersection = origin + dir * (intersect_distance - EPSILON);
  
  // Initialize the result color to the ambient light reflected in the shapes color
  vec result = intersected->get_color(intersection) * AMBIENT;
  
  // Add recursive reflections, unless we're at the recursion bound
  if(reflections < MAX_REFLECTIONS) {
  // Find the normal at the intersection point
  vec n = intersected->normal(intersection);

  // Reflect the vector across the normal
  vec new_dir = dir - n * 2.0 * n.dot(dir);
      
  // Compute the reflected color by recursively raytracing from this point
  vec reflected = raytrace(intersection, new_dir, reflections + 1);
  
  // Add the reflection to the result, tinted by the color of the shape
  result += reflected.hadamard(intersected->get_color(intersection)) *
  intersected->get_reflectivity();
    
  // Add the contribution from all lights in the scene
  for(vec& light : lights) {
  // Create a unit vector from the intersection to the light source
  vec shadow_dir = (light - intersection).normalized();

  // Check to see if the shadow vector intersects the scene
  bool in_shadow = false;
  for(shape* shape : scene) {
  if(shape->intersection(intersection, shadow_dir) >= 0) {
  in_shadow = true;
  break;
  }
  }
    
  // If there is a clear path to the light, add illumination
  if(!in_shadow) {
  // Compute the intensity of the diffuse lighting
  float diffuse_intensity = intersected->get_diffusion() *
  fmax(0, n.dot(shadow_dir));
      
  // Add diffuse lighting tinted by the color of the shape
  result += intersected->get_color(intersection) * diffuse_intensity;
        
  // Find the vector that bisects the eye and light directions
  vec bisector = (shadow_dir - dir).normalized();

  // Compute the intensity of the specular reflections, which are not affected by the color of the object
  float specular_intensity = intersected->get_spec_intensity() *
  fmax(0, pow(n.dot(bisector), (int)intersected->get_spec_density()));
      
  // Add specular highlights
  result += vec(1.0, 1.0, 1.0) * specular_intensity;
  }
  }
  } 
  return result; */
}

/**
 * Add objects and lights to the scene.
 * Creates three spheres, a flat plane, and two light sources
 */
void init_scene() {
  // Add a red sphere
  sphere* red_sphere = new sphere(vec(60, 50, 0), 50);
  red_sphere->set_color(vec(0.75, 0.125, 0.125));
  red_sphere->set_reflectivity(0.5);
  scene[0] = *red_sphere;
  
  // Add a green sphere
  sphere* green_sphere = new sphere(vec(-15, 25, -25), 25);
  green_sphere->set_color(vec(0.125, 0.6, 0.125));
  green_sphere->set_reflectivity(0.5);
  scene[1] = *green_sphere;
  
  // Add a blue sphere
  sphere* blue_sphere = new sphere(vec(-50, 40, 75), 40);
  blue_sphere->set_color(vec(0.125, 0.125, 0.75));
  blue_sphere->set_reflectivity(0.5);
  scene[2] = *blue_sphere;
  
  // Add a flat surface
  // plane* surface = new plane(vec(0, 0, 0), vec(0, 1, 0));
  // The following line uses C++'s lambda expressions to create a function
  /*
  surface->set_color([](vec pos) {
      // This function produces a grid pattern on the plane
      if((int)pos.x() % 100 == 0 || (int)pos.z() % 100 == 0) {
        return vec(0.3, 0.3, 0.3);
      } else {
        return vec(0.15, 0.15, 0.15);
      }
    });
  */ 
  //surface->set_diffusion(0.25);
  //surface->set_spec_density(10);
  //surface->set_spec_intensity(0.1);
  //scene[3] = *surface;
  
  // Add two lights
  lights[0] = vec(-1000, 300, 0);
  lights[1] = vec(100, 900, 500);
}
